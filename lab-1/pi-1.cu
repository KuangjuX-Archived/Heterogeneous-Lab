#include <stdio.h>
#include <hip/hip_runtime.h>

#include <time.h>
#include <sys/time.h>

const long double TimeConvert = 1e6;
#define BLOCK_NUM 32   //块数量
#define THREAD_NUM 256 // 每个块中的线程数
#define LOOP_N ((unsigned long long)(BLOCK_NUM * THREAD_NUM * 100000))

__global__ void integral_pi(double *g_sum) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    double tmp = 0;

    int idx = bid * THREAD_NUM + tid;
    int start = idx * 100000 + 1;
    int end = start + 100000;
    for(int i = start; i < end; i++) {
        tmp += 4. / (1 + ((i + 0.5) / LOOP_N) * ((i + 0.5) / LOOP_N));
    }
    g_sum[bid * THREAD_NUM + tid] = tmp;
}   

int main() {
    double *h_sum, *g_sum;
    double pi_v = 0;

    struct timeval start_time, end_time;
    gettimeofday(&start_time, NULL);

    // allocate host memory
    h_sum = (double*) malloc(sizeof(double) * BLOCK_NUM * THREAD_NUM);

    // Allocate device memory
    hipMalloc((void **)&g_sum, sizeof(double) * BLOCK_NUM * THREAD_NUM);

    // Execute kernels
    integral_pi<<<BLOCK_NUM,THREAD_NUM>>>(g_sum);

    // Transfer output from device memory to host
    hipMemcpy(h_sum, g_sum, sizeof(double)*BLOCK_NUM*THREAD_NUM, hipMemcpyDeviceToHost);

    for (int i = 0; i < BLOCK_NUM * THREAD_NUM; i++) {
        pi_v += h_sum[i];
    }

    gettimeofday(&end_time, NULL);
    long long int start, end;
    start = start_time.tv_sec * TimeConvert + start_time.tv_usec;
    end = end_time.tv_sec * TimeConvert + end_time.tv_usec;
    long double span_time;
    span_time = (end - start) / TimeConvert;
    printf("花费时间: %.6LFs.\n", span_time);

    printf("计算的 pi 值为： %.10f\n", (pi_v/LOOP_N));

    hipFree(g_sum);
    free(h_sum);

}