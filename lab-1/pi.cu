#include<stdio.h>
#include<hip/hip_runtime.h>


#define BLOCK_NUM 32   //块数量
#define THREAD_NUM 256 // 每个块中的线程数
#define LOOP_N BLOCK_NUM * THREAD_NUM * 1000000

__global__ void leib_pi(double* g_sum) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    double tmp = 0;
    int flag = -1;
    int idx = bid * THREAD_NUM + tid;
    int start = idx * 100000 + 1;
    int end = start + 100000;
    for (int i = start; i < end; i++) {
        tmp += flag * (1./(2 * i + 1));
        flag = -flag;
    }
    g_sum[bid*THREAD_NUM+tid] = tmp;
}

int main(){
    double *h_sum, *g_sum;
    double pi_v = 1;

    // allocate host memory
    h_sum = (double*) malloc(sizeof(double) * BLOCK_NUM * THREAD_NUM);

    // Allocate device memory
    hipMalloc((void **)&g_sum, sizeof(double) * BLOCK_NUM * THREAD_NUM);

    // Execute kernels
    leib_pi<<<BLOCK_NUM,THREAD_NUM>>>(g_sum);

    // Transfer output from device memory to host
    hipMemcpy(h_sum, g_sum, sizeof(double)*BLOCK_NUM*THREAD_NUM, hipMemcpyDeviceToHost);

    for (int i = 0; i < BLOCK_NUM * THREAD_NUM; i++) {
        pi_v += h_sum[i];
    }

    printf("calculate %.10f\n", pi_v*4);

    hipFree(g_sum);
    free(h_sum);
}